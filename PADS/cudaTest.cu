#include "hip/hip_runtime.h"
#include "cudaTest.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>

class Foo {
private:
	int x_;

public:
	__device__ Foo() { x_ = 42; }
	__device__ int bar() { return x_; }
};

__global__ void dvecTest(double *e){
	Foo f;
	e[0] = f.bar();
}

int cuMain() {
	int n;
	hipGetDevice(&n);

	double *he = (double *)malloc(sizeof(double));
	double *de;
	hipMalloc(&de, sizeof(double));

	dvecTest<<<1, 1>>>(de);

	hipMemcpy(he, de, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << *he << std::endl;
	return n;
}