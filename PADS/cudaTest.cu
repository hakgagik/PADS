#include "hip/hip_runtime.h"
#include "cudaTest.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <math.h>
#include "deviceVector.cuh"


__global__ void dvecTest(double *e){
	//double *a;
	//*a = 1;
	//dvec v(a, 1);
	//*e = v[0];

	double a[3];
	a[0] = 1; a[1] = 2; a[2] = 3;
	dvec d(a, 3);

	*e = d.norm();
}



int cuMain() {
	int n = 1;
	hipSetDevice(1);

	double *he = (double *)malloc(sizeof(double));
	double *de;
	hipMalloc(&de, sizeof(double));

	dvecTest<<<1, 1>>>(de);

	hipMemcpy(he, de, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << *he << std::endl;
	return n;
}