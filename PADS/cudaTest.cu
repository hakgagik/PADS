#include "hip/hip_runtime.h"
#include "cudaTest.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
//#include "deviceVector.cuh"
//

class Foo {
private:
	int x_;

public:
	__device__ Foo() { x_ = 42; }
	__device__ int bar() { return x_; }
};

__global__ void dvecTest(double *e){

	Foo f;
	*e = f.bar();
	//double *a;
	//*a = 1;
	//dvec v(a, 1);
	//*e = v[0];
}



int cuMain() {
	int n;

	double *he = (double *)malloc(sizeof(double));
	double *de;
	hipMalloc(&de, sizeof(double));

	dvecTest<<<1, 1>>>(de);

	hipMemcpy(he, de, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << *he << std::endl;
	return n;
}