#include "hip/hip_runtime.h"
#include "cudaTest.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include "deviceVector.cuh"

__global__ void dvecTest(double *e){
	double a[] = { 1.0, 2.0, 3.0 };
	int nElem = 3;

	dvec v(a, nElem);
	v += v;
	*e = v[1];
}

int cuMain() {
	int n;
	hipGetDevice(&n);
	return n;

	double *he = (double *)malloc(sizeof(double));
	double *de;
	hipMalloc(&de, sizeof(double));

	dvecTest<<<1, 1>>>(de);

	hipMemcpy(he, de, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << *he << std::endl;
}