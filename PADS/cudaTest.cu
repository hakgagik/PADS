#include "hip/hip_runtime.h"
#include "cudaTest.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <armadillo>
#include <cstdio>

__global__ void armaTest(double *e){
	arma::mat a(3, 3);
	arma::mat b(3, 3);
	a.fill(3);
	b.fill(3);
	arma::mat c = a * b;
	*e = c(1, 1);
}

int cuMain() {
	int n;
	hipGetDevice(&n);
	return n;

	double *he = (double *)malloc(sizeof(double));
	double *de;
	hipMalloc(&de, sizeof(double));
	hipMemcpy(he, de, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << *he << std::endl;
}