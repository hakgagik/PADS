#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "mainIterator.cuh"
#include <iostream>
#include <fstream>
#include <iomanip>

#define verletStride 100
#define cutoff 1.2
#define k_l 1.46E5
#define k_th 251.04
#define k_phi1 6.78
#define k_phi2 -3.6
#define k_phi3 13.56
#define l_0 0.153
#define th_0 1.187
#define eps 0.39
#define sigma 0.401
#define ax .40214
#define ay .11031
#define az .04552
#define bx 0.0
#define by .47345
#define bz .04051
#define cx -.00002849
#define cy -.00003859
#define cz 1.09983

// Each thread block contains nBeads threads. There are nMols thread blocks. Therefore, each molecule is its own thread block.
// Each molecule computes its own centroid and puts it into dcentroids
// This is done via a parallel reduction algorithm that calculates a sum of n elements in O(log(n)) time.
// Currently, this only works for n-alkanes where n is a power of 2
__global__ void getCentroids(double*x, double*y, double*z, double *dcentroidsx, double *dcentroidsy, double *dcentroidsz){
	int nBeads = blockDim.x;
	int i = blockIdx.x * nBeads + threadIdx.x;
	int level = 1;

	// Declare a shared variable to be used to calculate the centroid before copying into global memory.
	extern __shared__ double c[];
	double *mycx = c;
	double *mycy = &(c[nBeads]);
	double *mycz = &(c[2 * nBeads]);

	// Each bead copies its own info into the shared memory space.
	// Global memory bad!
	mycx[threadIdx.x] = x[i];
	mycy[threadIdx.x] = y[i];
	mycz[threadIdx.x] = z[i];

	__syncthreads();

	// Do a parallel reduction to find the sum of all the positions.
	while (level < nBeads){
		if (threadIdx.x % (2*level) == 0){
			mycx[threadIdx.x] += mycx[threadIdx.x + level];
			mycy[threadIdx.x] += mycy[threadIdx.x + level];
			mycz[threadIdx.x] += mycz[threadIdx.x + level];
		}
		level *= 2;
	}
	// Thread 0 then copies its own cx cy and cz back into global memory.
	if (threadIdx.x == 0){
		dcentroidsx[blockIdx.x] = mycx[0] / nBeads;
		dcentroidsy[blockIdx.x] = mycy[0] / nBeads;
		dcentroidsz[blockIdx.x] = mycz[0] / nBeads;
	}
}

// We next, we generate the Verlet list. This massively speeds up computation time.
// Now, each thread block has only one thread (representing one molecule). Each thread block will calculate its own verlet cell.
// This and the centroid calculation happen once every *very many* iterations because the verlet list is not very likely to change
// and centroids are only used to calculate the verlet list.
__global__ void getVerletList(int*verletList, int *verletListEnd, double*xCentroids, double*yCentroids, double*zCentroids, int nMols){
	// Copy own centroid into local memory and make local copies of all the parameters.
	double ctfsq = cutoff;
	ctfsq *= ctfsq;
	int mols = nMols;
	int stride = verletStride;
	int idx = blockIdx.x;
	double c[3];
	double dx[3];
	c[0] = xCentroids[idx];
	c[1] = yCentroids[idx];
	c[2] = zCentroids[idx];

	// Iterate forward, with 
	int verletCount = -1;
	for (int i = 1; i < mols; i++){
		int j = (i + idx) % mols;
		dx[0] = xCentroids[j] - c[0];
		dx[1] = yCentroids[j] - c[1];
		dx[2] = zCentroids[j] - c[2];
		if (dx[0] * dx[0] + dx[1] * dx[1] + dx[2] * dx[2] < ctfsq){
			verletCount++;
			verletList[stride * idx + verletCount] = j;
		}
	}
	if (verletCount >0){
		verletListEnd[idx] = stride * idx + verletCount;
	}
	else {
		verletListEnd[idx] = -1;
	}
}

// This is the main MD method.
// Technically, one octane (8 threads) per block is very inefficient. I should be using threadblocks of at least 32 threads. However, this would make programming a nightmare, as I'd have to first spend time figuring out how to organize the 4 octanes into memory and constantly making sure that they don't accidentally overlap.
__global__ void MDStep(double *xGlobal, double *yGlobal, double *zGlobal, int *verletList, int * verletListEnd, int nMols){
	// Copy constants into local memory... The caffeine in my bloodstream doesn't trust whatever's coming in through the functionc call >.>
	int i = blockIdx.x;
	int j = threadIdx.x;
	int b = blockDim.x;
	int m = nMols;

	// Pushing the max shared memory limit pretty hard here =/ Max shared memory is 49152B I'm using 19584B (although I should be able to cut this by half if things behave well enough) Right now, verletStride doesn't need to be more than 30.
	extern __shared__ double sharedMem[];
	double *x = sharedMem;
	double *y = &(sharedMem[b]);
	double *z = &(sharedMem[2 * b]);
	double *r = &(sharedMem[3 * b]);
	double *theta = &(sharedMem[4 * b]);
	double *phi = &(sharedMem[5 * b]);
	double *verletX = &(sharedMem[6 * b]);
	double *verletY = &(sharedMem[(6 + verletStride) * b]);
	double *verletZ = &(sharedMem[(6 + 2 * verletStride) * b]);

	// First, copy positions into shared memory. Each thread copies its own position. Thread = bead.
	x[j] = xGlobal[i*b + j];
	y[j] = yGlobal[i*b + j];
	z[j] = zGlobal[i*b + j];

	// Next, copy positions of other molecules in the verletList into shared memory. Each thread copies data corresponding to itself.
	int vCount = 0;
	for (int idx = verletStride * i; idx <= verletListEnd[i]; idx++){
		verletX[vCount * b + j] = xGlobal[verletList[idx] * b + j];
		verletY[vCount * b + j] = yGlobal[verletList[idx] * b + j];
		verletZ[vCount * b + j] = zGlobal[verletList[idx] * b + j];
		vCount++;
	}

	__syncthreads();

	// Each molecule stores a vector to the next bead and the previous bead.
	// The third set of beads is the distance between the next bead and the bead after that (for torsion calculation).
	double dxp, dyp, dzp, dxm, dym, dzm, dxpp, dypp, dzpp;

	if (j < (b - 1)) {
		dxp = x[j + 1] - x[j];
		dyp = y[j + 1] - y[j];
		dzp = z[j + 1] - z[j];
	}
	else {
		dxp = 0;
		dyp = 0;
		dzp = 0;
	}

	if (j > 0){
		dxm = x[j - 1] - x[j];
		dym = y[j - 1] - y[j];
		dzm = z[j - 1] - z[j];
	}
	else {
		dxm = 0;
		dym = 0;
		dzm = 0;
	}

	if (j < (b - 2)) {
		dxpp = x[j + 2] - x[j + 1];
		dypp = y[j + 2] - y[j + 1];
		dzpp = z[j + 2] - z[j + 1];
	}

	// Next, calculate distances between beads. Each thread calculates the distance to the next bead.
	r[j] = sqrt(dxp * dxp + dyp * dyp + dzp * dzp);

	__syncthreads();

	// Now, calculate angles between beads. Each bead claculates the angle that has it at the origin.
	// The threads at the edges will produce nonsensical results, but we're not going to access them (in any useful manner) anyway.
	// Allowing the edge threads to calculate angles minimizes thread divergence (different theads doing different things).
	theta[j] = acos((dxp * dxm + dyp * dym + dzp * dzm) / r[j] / r[j - 1]);

	// Finally, calculate four-molecule torsion angles.
	// Same as above, this will produce some nonsensical data, but we won't be accessing it.
	phi[j] = acos((dxm * dxpp + dym * dypp + dzm * dzpp) / r[j - 1] / r[j + 1]);

	// Now, each molecule calculates a force on itself from ALL the terms. ALL OF THEM.

	double Fx = 0, Fy = 0, Fz = 0;

	// First, the spring term. Each bead receives a contribution from the bead ahead of it and from the bead behind it.
	double factor;
	if (j > 0) {
		factor = 2 * k_l * (l_0 - r[j]) / r[j];
		Fx += factor * dxp;
		Fy += factor * dyp;
		Fz += factor * dzp;
	}
	if (j < (b - 1)) {
		factor = 2 * k_l * (l_0 - r[j - 1]) / r[j - 1];
		Fx += factor * dxm;
		Fy += factor * dym;
		Fz += factor * dzm;
	}

	// Next, the theta term. A bit more complicated. Each molecule recieve a contribution from the angle behind it, the angle ahead of it, and the angle that has it as the origin.
	if (j < (b - 2)) {
		factor = 2 * k_th * (th_0 - theta[j + 1])/r[j];
		Fx += factor * (dxpp / r[j + 1] + cos(theta[j + 1]) * dxp / r[j]);
		Fy += factor * (dypp / r[j + 1] + cos(theta[j + 1]) * dyp / r[j]);
		Fz += factor * (dzpp / r[j + 1] + cos(theta[j + 1]) * dzp / r[j]);
	}
	if (j > 1) {
		factor = 2 * k_th * (th_0 - theta[j - 1] / r[j - 1]);
		Fx += factor * ((x[j - 2] - x[j - 1]) / r[j - 2] + cos(theta[j - 1]) * dxm);
		Fy += factor * ((y[j - 2] - y[j - 1]) / r[j - 2] + cos(theta[j - 1]) * dym);
		Fz += factor * ((z[j - 2] - z[j - 1]) / r[j - 2] + cos(theta[j - 1]) * dzm);
	}
	if (j > 0 && j < (b - 1)){
		factor = 2 * k_th * (th_0 - theta[j]);
		Fx -= factor * (dxm / r[j - 1] - cos(theta[j]) * dxp / r[j]) / r[j]
			+ (dxp / r[j] - cos(theta[j]) * dxm / r[j - 1]) / r[j - 1];
	}

	// Next phi. WHAT. THE. FUCK. Is WRONG with GIT?!
	if (j < (b - 3)){
		factor = 0.5 * (k_phi1 * sin(phi[j + 1]) + 2 * k_phi2 * sin(2 * phi[j + 1]) + 3 * k_phi3 * sin(3 * phi[j + 1])) / sin(phi[j + 1]);
		Fx += factor * ((x[j + 3] - x[j + 2]) / r[j + 2] + cos(phi[j + 1]) * dxp / r[j]) / r[j];
		Fy += factor * ((y[j + 3] - y[j + 2]) / r[j + 2] + cos(phi[j + 1]) * dyp / r[j]) / r[j];
		Fz += factor * ((z[j + 3] - z[j + 2]) / r[j + 2] + cos(phi[j + 1]) * dzp / r[j]) / r[j];
	}
	if (j < (b - 2) && j > 0) {
		factor = 0.5 * (k_phi1 * sin(phi[j]) + 2 * k_phi2 * sin(2 * phi[j]) + 3 * k_phi3 * sin(3 * phi[j])) / sin(phi[j]);
		Fx -= factor * (dxpp / r[j + 1] - cos(phi[j]) * dxm / r[j - 1]) / r[j - 1];
		Fy -= factor * (dypp / r[j + 1] - cos(phi[j]) * dym / r[j - 1]) / r[j - 1];
		Fz -= factor * (dzpp / r[j + 1] - cos(phi[j]) * dzm / r[j - 1]) / r[j - 1];
	}
	if (j < (b - 1) && j > 1) {
		factor = 0.5 * (k_phi1 * sin(phi[j - 1]) + 2 * k_phi2 * sin(2 * phi[j - 1]) + 3 * k_phi3 * sin(3 * phi[j - 1])) / sin(phi[j - 1]);
		Fx -= factor * ((x[j - 2] - x[j - 1]) / r[j - 2] - cos(phi[j - 1]) * dxp / r[j]) / r[j];
		Fy -= factor * ((y[j - 2] - y[j - 1]) / r[j - 2] - cos(phi[j - 1]) * dyp / r[j]) / r[j];
		Fz -= factor * ((z[j - 2] - z[j - 1]) / r[j - 2] - cos(phi[j - 1]) * dzp / r[j]) / r[j];
	}
	if (j > 2) {
		factor = 0.5 * (k_phi1 * sin(phi[j - 2]) + 2 * k_phi2 * sin(2 * phi[j - 2]) + 3 * k_phi3 * sin(3 * phi[j - 2])) / sin(phi[j - 2]);
		Fx += factor * ((x[j - 3] - x[j - 2]) / r[j - 3] + cos(phi[j - 2]) * dxm / r[j - 1]) / r[j - 1];
		Fy += factor * ((y[j - 3] - y[j - 2]) / r[j - 3] + cos(phi[j - 2]) * dym / r[j - 1]) / r[j - 1];
		Fz += factor * ((z[j - 3] - z[j - 2]) / r[j - 3] + cos(phi[j - 2]) * dzm / r[j - 1]) / r[j - 1];
	}

}


int cuMainLoop(double *x, double *y, double *z, int nMols, int nBeads){
	
	hipSetDevice(1);

	// d in front of a variable in this functions means it's a device variable
	double *dx, *dy, *dz;

	double *dcentroidsx, *dcentroidsy, *dcentroidsz;

	hipMalloc(&dcentroidsx, sizeof(double)*nMols);
	hipMalloc(&dcentroidsy, sizeof(double)*nMols);
	hipMalloc(&dcentroidsz, sizeof(double)*nMols);

	hipMalloc(&dx, sizeof(double) * nBeads * nMols);
	hipMalloc(&dy, sizeof(double) * nBeads * nMols);
	hipMalloc(&dz, sizeof(double) * nBeads * nMols);

	hipMemcpy(dx, x, nMols * nBeads * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dy, y, nMols * nBeads * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dz, z, nMols * nBeads * sizeof(double), hipMemcpyHostToDevice);

	getCentroids<<<nMols, nBeads, 3 * nBeads * sizeof(double)>>>(dx, dy, dz, dcentroidsx, dcentroidsy, dcentroidsz);

	int* verletList;
	int* verletListEnd;


	hipMalloc(&verletList, sizeof(int) * nMols * verletStride);
	hipMalloc(&verletListEnd, sizeof(int) * nMols);

	getVerletList<<<nMols,1>>>(verletList, verletListEnd, dcentroidsx, dcentroidsy, dcentroidsz, nMols);

	MDStep<<<nMols,nBeads, (3 * verletStride * nBeads + 6 * nBeads) * sizeof(double)>>>(dx, dy, dz, verletList, verletListEnd, nMols);

	return EXIT_SUCCESS;
}